#include "hip/hip_runtime.h"
// Copyright (c) 2020 Michael Koesel and respective contributors
// SPDX-License-Identifier: MIT
// See accompanying LICENSE file for detailed information

#include "dogm/common.h"
#include "dogm/cuda_utils.h"
#include "dogm/dogm_types.h"
#include "dogm/kernel/ego_motion_compensation.h"

#include <hip/hip_runtime.h>
#include <>

namespace dogm
{

__global__ void moveParticlesKernel(ParticlesSoA particle_array, float x_move, float y_move,
                                    float cos_theta, float sin_theta, int particle_count)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < particle_count; i += blockDim.x * gridDim.x)
    {
        // TODO: which way should the state move here?  For some reason it was opposite in direction to the
        //       movement of the map kernel below.
        particle_array.state[i][0] = int(  cos_theta * particle_array.state[i][0]
                                         + sin_theta * particle_array.state[i][1]
                                         - x_move);
        particle_array.state[i][1] = int( -sin_theta * particle_array.state[i][0]
                                         + cos_theta * particle_array.state[i][1]
                                         - y_move);
    }
}

__global__ void moveMapKernel(GridCell* __restrict__ grid_cell_array, const GridCell* __restrict__ old_grid_cell_array,
                              float cos_theta, float sin_theta, float x_move, float y_move, int grid_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < grid_size && y < grid_size)
    {
        int index = x + grid_size * y;
        // TODO: same as above -- x and y translation should be negative?  Currently opposite the particles fn above
        int new_x = int(cos_theta * x + sin_theta * y + x_move);
        int new_y = int(- sin_theta * x + cos_theta * y + y_move);
        int new_index = new_x + grid_size * new_y;

        if (new_x > 0 && new_x < grid_size && new_y > 0 && new_y < grid_size)
        {
            grid_cell_array[index] = old_grid_cell_array[new_index];
        }
    }
}

} /* namespace dogm */
