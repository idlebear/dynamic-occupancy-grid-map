#include "hip/hip_runtime.h"
// Copyright (c) 2020 Michael Koesel and respective contributors
// SPDX-License-Identifier: MIT
// See accompanying LICENSE file for detailed information

#include "dogm/common.h"
#include "dogm/cuda_utils.h"
#include "dogm/dogm_types.h"
#include "dogm/kernel/ego_motion_compensation.h"

#include <hip/hip_runtime.h>
#include <>

namespace dogm
{

__global__ void moveParticlesKernel(ParticlesSoA particle_array, float x_move, float y_move,
                                    float cos_theta, float sin_theta, int particle_count)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < particle_count; i += blockDim.x * gridDim.x)
    {
        auto x = particle_array.state[i][0];
        auto y = particle_array.state[i][1];
        auto vx = particle_array.state[i][2];
        auto vy = particle_array.state[i][3];

        particle_array.state[i][0] = cos_theta * x + sin_theta * y - x_move;
        particle_array.state[i][1] = -sin_theta * x + cos_theta * y - y_move;
        particle_array.state[i][2] = cos_theta * vx + sin_theta * vy;
        particle_array.state[i][3] = -sin_theta * vx + cos_theta * vy;
    }
}

__global__ void moveMapKernel(GridCell* __restrict__ grid_cell_array, const GridCell* __restrict__ old_grid_cell_array,
                              float cos_theta, float sin_theta, float x_move, float y_move, int grid_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < grid_size && y < grid_size)
    {
        int new_x = int(cos_theta * float(x) + sin_theta * float(y) - x_move);
        int new_y = int(-sin_theta * float(x) + cos_theta * float(y) - y_move);

        if (new_x >= 0 && new_x < grid_size && new_y >= 0 && new_y < grid_size)
        {
            int index = x + grid_size * y;
            int new_index = new_x + grid_size * new_y;
            grid_cell_array[new_index] = old_grid_cell_array[index];

            // rotate the mean cell velocities as well
            // TODO: May also need to 'rotate' the variances as well, but since they are always positive, it
            //       really amounts to a reproportioning.
            auto mean_x_vel = grid_cell_array[new_index].mean_x_vel;
            auto mean_y_vel = grid_cell_array[new_index].mean_y_vel;

            grid_cell_array[new_index].mean_x_vel = cos_theta * mean_x_vel + sin_theta * mean_y_vel;
            grid_cell_array[new_index].mean_y_vel = -sin_theta * mean_x_vel + cos_theta * mean_y_vel;
        }
    }
}

} /* namespace dogm */
