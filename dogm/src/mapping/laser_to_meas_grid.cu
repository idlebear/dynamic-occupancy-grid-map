#include "hip/hip_runtime.h"
// Copyright (c) 2020 Michael Koesel and respective contributors
// SPDX-License-Identifier: MIT
// See accompanying LICENSE file for detailed information

#include "dogm/mapping/laser_to_meas_grid.h"
#include "dogm/mapping/kernel/measurement_grid.h"

dogm::LaserMeasurementGrid::LaserMeasurementGrid(const Params& laser_params, float grid_length, float grid_resolution)
    : grid_size(static_cast<int>(grid_length / grid_resolution)), grid_resolution( grid_resolution), laser_params(laser_params),
      polar_width(ceil( laser_params.fov / laser_params.angle_increment )), polar_height( ceil( laser_params.max_range / laser_params.resolution ) )
{
    int grid_cell_count = grid_size * grid_size;

    meas_grid.init(grid_cell_count, true);

    theta_min = - (laser_params.fov / 2.0);
    CUDA_CALL(hipMalloc(&polar_grid, polar_width * polar_height * sizeof(float2)));
}

dogm::LaserMeasurementGrid::~LaserMeasurementGrid()
{
    CUDA_CALL(hipFree(polar_grid));
    meas_grid.free();
}

dogm::MeasurementCellsSoA dogm::LaserMeasurementGrid::generateGrid(const std::vector<float>& measurements)
{
    const int num_measurements = measurements.size();

    float* d_measurements;
    CUDA_CALL(hipMalloc(&d_measurements, num_measurements * sizeof(float)));
    CUDA_CALL(
        hipMemcpy(d_measurements, measurements.data(), num_measurements * sizeof(float), hipMemcpyHostToDevice));

    dim3 dim_block(32, 32);
    dim3 polar_grid_dim(divUp(polar_width, dim_block.x), divUp(polar_height, dim_block.y));
    dim3 cart_grid_dim(divUp(grid_size, dim_block.x), divUp(grid_size, dim_block.y));

    // convert the measurement information into a polar representation
    createPolarGridKernel<<<polar_grid_dim, dim_block>>>(polar_grid, d_measurements, polar_width, polar_height,
                                                          laser_params.resolution);

    CUDA_CALL(hipGetLastError());

    // // transform polar representation to a cartesian grid
    transformPolarGridToCartesian<<<cart_grid_dim, dim_block>>>( meas_grid, grid_size, grid_resolution,
        polar_grid, polar_width, polar_height, theta_min, laser_params.angle_increment, laser_params.resolution,
        true );
    CUDA_CALL(hipGetLastError());

    CUDA_CALL(hipFree(d_measurements));
    CUDA_CALL(hipDeviceSynchronize());

    return meas_grid;
}
